#include "hip/hip_runtime.h"
#include "../operator_matrix.cuh"
#include "cutlass_gemm_op.cuh"
#include <chrono>


void i8_perf_test(){

    int test_para[2048][3] = {
        {1024,1024,1024},
        {2048,2048,2048},
        {3072,3072,3072},
        {4096,4096,4096},
        {5120,5120,5120},
        {6144,6144,6144},
        {7168,7168,7168},
        {8192,8192,8192},
        {9216,9216,9216},
        {10240,10240,10240},
        {11264,11264,11264},
        {12288,12288,12288},
        {13312,13312,13312},
        {14336,14336,14336},
        {15360,15360,15360},
        {16384,16384,16384},
        {17408,17408,17408},
        {18432,18432,18432},
        {19456,19456,19456},
        {20480,20480,20480},
        {21504,21504,21504},
        {22528,22528,22528},
        {23552,23552,23552},
        {24576,24576,24576},
        {25600,25600,25600},
        {26624,26624,26624},
        {27648,27648,27648},
        {28672,28672,28672},
        {29696,29696,29696},
        {30720,30720,30720},
        {31744,31744,31744},
        {32768,32768,32768},
    }; 

    // 定义数组的大小

    int max = 4096*8;
    // 创建一个使用float类型的数组
    std::vector<int8_t> int4b_arrayA(max*max);
    std::vector<int8_t> int4b_arrayB(max*max);
    std::vector<int32_t> int32b_arrayC(max*max);

    int8_t* d_A;
    int8_t* d_B;
    int32_t* d_C;
    hipMalloc((void**)&d_A, sizeof(int8_t) * max*max);
    hipMalloc((void**)&d_B, sizeof(int8_t) * max*max);
    hipMalloc((void**)&d_C, sizeof(int32_t) * max*max);
    hipMemcpy(d_A, int4b_arrayA.data(), sizeof(int8_t) * max*max, hipMemcpyHostToDevice);
    hipMemcpy(d_B, int4b_arrayB.data(), sizeof(int8_t) * max*max, hipMemcpyHostToDevice);

    float beta = 0.0, alpha = 1.0;

    for(int i=0;i<32;i++){
        

        int N=test_para[i][0],M=test_para[i][1],K=test_para[i][2];
        float alpha = 1.0, beta = 0.0;
        if(M==0) return;

        std::cout<<M<<"\t"<<N<<"\t"<<K<<"\t";

        //计算float和int矩阵乘法得到结果矩阵
        cut_gemm(d_A, d_B, d_C,M,K, K,N);
        hipDeviceSynchronize();
        {
            auto start = std::chrono::high_resolution_clock::now();
            cut_gemm(d_A, d_B, d_C,M,K, K,N);
            hipDeviceSynchronize();
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double, std::milli> diff = end - start;
            double time  = diff.count();
            printf("%.7lf\n",time);
            hipDeviceSynchronize();
        }

    }
    return;        

}

void i8_acc_test() {
    // 定义数组的大小
    int M=1024,N=1024,K=512;
    // 创建一个使用input_t类型的数组
    std::vector<input_t> int4b_arrayA(M*K);
    std::vector<input_t> int4b_arrayB(K*N);
    std::vector<int32_t> int32b_arrayC(M*N);


    //初始化数组
    for (int i = 0; i < M; ++i) {

        for(int j=0;j<K;j++){
            // 将每个元素初始化为它的索引值，注意这里只是示例，实际值可能需要根据量化规则来确定
            // if(i==0||i==1)int4b_arrayA[i*K+j] = static_cast<input_t>(i+1);
            // if(j==0||j==1)int4b_arrayB[i+j*K] = static_cast<input_t>(j+1);
            int4b_arrayA[i*K+j] = static_cast<input_t>(1);
        }

    }


    for (int i = 0; i < K; ++i) {
        for(int j=0;j< N;j++){
            int4b_arrayB[i*N+j] = static_cast<input_t>(2);
        }
    }
    for (int i = 0; i < M; ++i) {
        for(int j=0;j< N;j++){
            int32b_arrayC[i*N+j] = 0;
        }
    }
    // for (int i = 0; i < M; ++i) {
    //     for(int j=0;j<K;j++){
    //         printf("%d,",static_cast<int>(int4b_arrayA[i*M+j]));
    //     }
    //     printf("\n");
    // }
    // printf("\n");
    // for (int i = 0; i < K; ++i) {
    //     for(int j=0;j< N;j++){
    //         printf("%d,",static_cast<int>(int4b_arrayB[i*M+j]));
    //     }
    //     printf("\n");
    // }
    // printf("\n");

    input_t* d_A;
    input_t* d_B;
    input_t* d_B_TP;
    int32_t* d_C;
    hipMalloc((void**)&d_A, sizeof(input_t) * M*N);
    hipMalloc((void**)&d_B, sizeof(input_t) * M*N);
    hipMalloc((void**)&d_C, sizeof(int32_t) * M*N);
    hipMalloc((void**)&d_B_TP, sizeof(int32_t) * M*N);
    hipMemcpy(d_A, int4b_arrayA.data(), sizeof(input_t) * M*N, hipMemcpyHostToDevice);
    hipMemcpy(d_B, int4b_arrayB.data(), sizeof(input_t) * M*N, hipMemcpyHostToDevice);

    I8trans(d_B_TP,d_B,K,N);
    cut_gemm(d_A, d_B_TP, d_C,M,K, K,N);
    hipDeviceSynchronize();
    
    hipMemcpy( int32b_arrayC.data(),d_C, sizeof(int32_t) * M*N, hipMemcpyDeviceToHost);
  
    for (int i = 0; i < 32; ++i) {
        for(int j=0;j< 32;j++){
            printf("%d,",static_cast<int>(int32b_arrayC[i*M+j]));
        }
        printf("\n");
    }





    return ;
}

int main(){
    i8_perf_test();
}