#include "../operator_matrix.cuh"
#include "cutlass_gemm_op.cuh"



int main() {
    // 定义数组的大小
    int M=32,N=32,K=32;
    // 创建一个使用input_t类型的数组
    std::vector<input_t> int4b_arrayA(M*K);
    std::vector<input_t> int4b_arrayB(K*N);
    std::vector<int32_t> int32b_arrayC(K*N);


    // 初始化数组
    for (int i = 0; i < M; ++i) {

        for(int j=0;j<K;j++){
            // 将每个元素初始化为它的索引值，注意这里只是示例，实际值可能需要根据量化规则来确定
            if(i==0||i==1)int4b_arrayA[i*K+j] = static_cast<input_t>(i+1);
            if(j==0||j==1)int4b_arrayB[i*K+j] = static_cast<input_t>(j+1);
            int32b_arrayC[i*K+j] = 0;
        }

    }

    for (int i = 0; i < M; ++i) {
        for(int j=0;j<N;j++){
            printf("%d,",static_cast<int>(int4b_arrayA[i*M+j]));
        }
        printf("\n");
    }
    printf("\n");
    for (int i = 0; i < M; ++i) {
        for(int j=0;j<N;j++){
            printf("%d,",static_cast<int>(int4b_arrayB[i*M+j]));
        }
        printf("\n");
    }
    printf("\n");

    input_t* d_A;
    input_t* d_B;
    int32_t* d_C;
    hipMalloc((void**)&d_A, sizeof(input_t) * M*N);
    hipMalloc((void**)&d_B, sizeof(input_t) * M*N);
    hipMalloc((void**)&d_C, sizeof(int32_t) * M*N);
    hipMemcpy(d_A, int4b_arrayA.data(), sizeof(input_t) * M*N, hipMemcpyHostToDevice);
    hipMemcpy(d_B, int4b_arrayB.data(), sizeof(input_t) * M*N, hipMemcpyHostToDevice);

    I8trans(d_B,d_B,K,N);
    cut_gemm(d_A, d_B, d_C,M,K, K,N);

    
    hipMemcpy( int32b_arrayC.data(),d_C, sizeof(int32_t) * M*N, hipMemcpyDeviceToHost);
  
    for (int i = 0; i < M; ++i) {

        for(int j=0;j<N;j++){
            printf("%d,",static_cast<int>(int32b_arrayC[i*M+j]));
        }
        printf("\n");
    }



    return 0;
}