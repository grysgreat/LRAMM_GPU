



#include "cutlass_gemm_op.cuh"



int main() {
    // 定义数组的大小
    int M=512,N=512,K=512;
    // 创建一个使用input_t类型的数组
    std::vector<input_t> int4b_arrayA(M*K);
    std::vector<input_t> int4b_arrayB(K*N);
    std::vector<int32_t> int32b_arrayC(K*N);


    // 初始化数组
    for (int i = 0; i < M; ++i) {

        for(int j=0;j<K;j++){
            // 将每个元素初始化为它的索引值，注意这里只是示例，实际值可能需要根据量化规则来确定
            int4b_arrayA[i*K+j] = static_cast<input_t>(i);
            int4b_arrayB[i*K+j] = static_cast<input_t>(i);
            int32b_arrayC[i*K+j] = 0;
        }

    }

    input_t* d_A;
    input_t* d_B;
    int32_t* d_C;
    hipMalloc((void**)&d_A, sizeof(input_t) * M*N);
    hipMalloc((void**)&d_B, sizeof(input_t) * M*N);
    hipMalloc((void**)&d_C, sizeof(int32_t) * M*N);
    hipMemcpy(d_A, int4b_arrayA.data(), sizeof(input_t) * M*N, hipMemcpyHostToDevice);
    hipMemcpy(d_B, int4b_arrayB.data(), sizeof(input_t) * M*N, hipMemcpyHostToDevice);


    cut_gemm(d_A, d_B, d_C,M,K, K,N);

    
    hipMemcpy( int32b_arrayC.data(),d_C, sizeof(int32_t) * M*N, hipMemcpyDeviceToHost);
  
    for (int i = 0; i < 10; ++i) {

        for(int j=0;j<10;j++){
            printf("%d,",static_cast<int>(int32b_arrayC[i*M+j]));
        }
        printf("\n");
    }



    return 0;
}