#include "hip/hip_runtime.h"
#include "../cublas/cublas_lib.cuh"
#include "../operator_matrix.cuh"
#include "../gen_matrix.cuh"

#include <iomanip>
#include "stdio.h"
#include <chrono>


template <typename T>
T get_Ferror(T matrix_ref[],T matrix_cmp[],int rows,int cols){

    T sumR=0,sum=0;
    for(int i=0;i<rows;i++){
        for(int j=0;j<cols;j++){
            sumR+=(matrix_ref[i*cols+j] - matrix_cmp[i*cols+j])*(matrix_ref[i*cols+j] - matrix_cmp[i*cols+j]);
            sum+=(matrix_ref[i*cols+j])*(matrix_ref[i*cols+j]);
        }
    }

    T ans = sqrt(sumR)/sqrt(sum);
    return ans;

}


int axpy_perf_test(){
    int num = 8192;
    int N=num,M=num,K=num;

    float *matrixA = (float *)malloc(sizeof(float) * M*N);
    int8_t *matrixA8 = (int8_t *)malloc(sizeof(int8_t) * M*N);
    float *matrixB = (float *)malloc(sizeof(float) * M*N);

    float *vec_row = (float *)malloc(sizeof(float) * M*N);
    float *vec_col = (float *)malloc(sizeof(float) * M*N);
    float *work = (float *)malloc(sizeof(float) * M*N);
    generate_matrix<float>(matrixA,M,N,'u');
    generate_matrix<float>(matrixB,M,N,'u');

    float *matrixA_dev;
    float *matrixB_dev;
    float *work_dev;
    hipMalloc((void**)&matrixA_dev, sizeof(float) * M*N);
    hipMalloc((void**)&matrixB_dev, sizeof(float) * M*N);

    // start = std::chrono::high_resolution_clock::now();
    hipMemcpy(matrixA_dev, matrixA, sizeof(float) * M*N, hipMemcpyHostToDevice);
    hipMemcpy(matrixB_dev, matrixA, sizeof(float) * M*N, hipMemcpyHostToDevice);

    float alpha = 1.0;

    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;
    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));


    auto start = std::chrono::high_resolution_clock::now();
    cublas_saxpy(matrixA_dev,matrixB_dev,alpha, num*num,cublasH,stream);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = end - start;
    int time  = diff.count()*1000*1000;
    std::cout<<"size="<<M<<"//axpy - gpu time:" << std::fixed << std::setprecision(6) << time << std::endl;


    hipMemcpy(matrixB, matrixB_dev, sizeof(float) * M*N, hipMemcpyDeviceToHost);
}


void gemm_acc_test(){
    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    // 定义数组的大小
    int M=16,N=14,K=2;
    // 创建一个使用float类型的数组
    std::vector<float> int4b_arrayA(M*K);
    std::vector<float> int4b_arrayB(K*N);
    std::vector<float> int32b_arrayC(M*N);


    // 初始化数组
    for (int i = 0; i < M; ++i) {
        for(int j=0;j<K;j++){
            // 将每个元素初始化为它的索引值，注意这里只是示例，实际值可能需要根据量化规则来确定
            int4b_arrayA[i*K+j] = static_cast<float>(i*K+j);
        }
    }
    for (int i = 0; i < K; ++i) {
        for(int j=0;j<N;j++){
            // 将每个元素初始化为它的索引值，注意这里只是示例，实际值可能需要根据量化规则来确定
            int4b_arrayB[i*N+j] = static_cast<float>(j);
        }
    }


    for (int i = 0; i < M; ++i) {
        for(int j=0;j<K;j++){
            printf("%d,",static_cast<int>(int4b_arrayA[i*K+j]));
        }
        printf("\n");
    }
    printf("\n");
    for (int i = 0; i < K; ++i) {
        for(int j=0;j<N;j++){
            printf("%d,",static_cast<int>(int4b_arrayB[i*N+j]));
        }
        printf("\n");
    }
    printf("\n");

    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc((void**)&d_A, sizeof(float) * M*K);
    hipMalloc((void**)&d_B, sizeof(float) * K*N);
    hipMalloc((void**)&d_C, sizeof(float) * M*N);
    hipMemcpy(d_A, int4b_arrayA.data(), sizeof(float) * M*K, hipMemcpyHostToDevice);
    hipMemcpy(d_B, int4b_arrayB.data(), sizeof(float) * K*N, hipMemcpyHostToDevice);

    float  beta = 0.0, alpha = 1.0;
    // hipblasSgemm(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K,
    //             &alpha, d_A, M, d_B, M, &beta, d_C, M);  

    // hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K,
    //             &alpha, d_B, N, d_A, K, &beta, d_C, N);  

    cublas_gemm_rowmajor(
        &cublasH, d_A, d_B, d_C, M, K,
        K, N, alpha, beta);

    hipMemcpy( int32b_arrayC.data(),d_C, sizeof(float) * M*N, hipMemcpyDeviceToHost);
  
    for (int i = 0; i < M; ++i) {
        for(int j=0;j<N;j++){
            printf("%d,",static_cast<int>(int32b_arrayC[i*N+j]));
        }
        printf("\n");
    }

}


void gemm_acc_test2(){
    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    // 定义数组的大小
    int M=16,N=14,K=2;
    // 创建一个使用float类型的数组
    std::vector<float> int4b_arrayA(M*K);
    std::vector<float> int4b_arrayB(K*N);
    std::vector<float> int32b_arrayC(M*N);


    // 初始化数组
    for (int i = 0; i < M; ++i) {
        for(int j=0;j<K;j++){
            // 将每个元素初始化为它的索引值，注意这里只是示例，实际值可能需要根据量化规则来确定
            int4b_arrayA[i*K+j] = static_cast<float>(i*K+j);
        }
    }
    for (int i = 0; i < K; ++i) {
        for(int j=0;j<N;j++){
            // 将每个元素初始化为它的索引值，注意这里只是示例，实际值可能需要根据量化规则来确定
            int4b_arrayB[i*N+j] = static_cast<float>(j);
        }
    }


    for (int i = 0; i < M; ++i) {
        for(int j=0;j<K;j++){
            printf("%d,",static_cast<int>(int4b_arrayA[i*K+j]));
        }
        printf("\n");
    }
    printf("\n");
    for (int i = 0; i < K; ++i) {
        for(int j=0;j<N;j++){
            printf("%d,",static_cast<int>(int4b_arrayB[i*N+j]));
        }
        printf("\n");
    }
    printf("\n");

    float* d_A;
    float* d_B;
    float* d_C;
    float* d_C_TMP;
    hipMalloc((void**)&d_A, sizeof(float) * M*K);
    hipMalloc((void**)&d_B, sizeof(float) * K*N);
    hipMalloc((void**)&d_C, sizeof(float) * M*N);
    hipMalloc((void**)&d_C_TMP, sizeof(float) * M*N);
    hipMemcpy(d_A, int4b_arrayA.data(), sizeof(float) * M*K, hipMemcpyHostToDevice);
    hipMemcpy(d_B, int4b_arrayB.data(), sizeof(float) * K*N, hipMemcpyHostToDevice);

    float  beta = 0.0, alpha = 1.0;
    // hipblasSgemm(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K,
    //             &alpha, d_A, M, d_B, M, &beta, d_C, M);  

    // hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K,
    //             &alpha, d_B, N, d_A, K, &beta, d_C, N);  


    hipblasSgemm(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, M, N, K, &alpha, d_A, K, d_B, K, &beta, d_C_TMP, M);
    hipblasSgeam(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, N, M, &alpha, d_C_TMP, M, &beta, d_C, N, d_C, N);
    hipDeviceSynchronize();


    hipMemcpy( int32b_arrayC.data(),d_C, sizeof(float) * M*N, hipMemcpyDeviceToHost);
  
    for (int i = 0; i < M; ++i) {
        for(int j=0;j<N;j++){
            printf("%d,",static_cast<int>(int32b_arrayC[i*N+j]));
        }
        printf("\n");
    }

}


void gemv_acc_test(){
    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    // 定义数组的大小
    int M=32,N=16;
    // 创建一个使用float类型的数组
    std::vector<float> int4b_arrayA(M*N);
    std::vector<float> int4b_arrayB(N);
    std::vector<float> int32b_arrayC(M*4);


    // 初始化数组
    for (int i = 0; i < M; ++i) {
        for(int j=0;j<N;j++){
            // 将每个元素初始化为它的索引值，注意这里只是示例，实际值可能需要根据量化规则来确定
            int4b_arrayA[i*N+j] = static_cast<float>(i+1);
        }
    }
    for (int i = 0; i < N; ++i) int4b_arrayB[i] = i;
    for (int i = 0; i < M; ++i) int32b_arrayC[i] = 0;

    



    for (int i = 0; i < M; ++i) {
        for(int j=0;j<N;j++){
            printf("%d,",static_cast<int>(int4b_arrayA[i*N+j]));
        }
        printf("\n");
    }
    printf("\n");
    for (int i = 0; i < N; ++i) printf("%d,",static_cast<int>(int4b_arrayB[i]));

    printf("\n");

    float* d_A;
    float* d_B;
    float* d_C;
    float* d_A_tmp;
    hipMalloc((void**)&d_A, sizeof(float) * M*N);
    hipMalloc((void**)&d_A_tmp, sizeof(float) * M*N);
    hipMalloc((void**)&d_B, sizeof(float) * N);
    hipMalloc((void**)&d_C, sizeof(float) * M);
    hipMemcpy(d_A, int4b_arrayA.data(), sizeof(float) * M*N, hipMemcpyHostToDevice);
    hipMemcpy(d_B, int4b_arrayB.data(), sizeof(float) * N, hipMemcpyHostToDevice);

    float  beta = 0.0, alpha = 1.0;
    
    // strans(d_A_tmp,d_A,M,N);
    // hipblasSgemv(cublasH, HIPBLAS_OP_N, M, N, 
    //             &alpha, d_A_tmp, M, d_B, 1, &beta, d_C, 1);  

    hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, 4, 4,
                    &alpha, d_B, 1, d_A, N, &beta, d_C, 4);  
    hipDeviceSynchronize();

    hipMemcpy( int32b_arrayC.data(),d_C, sizeof(float) * M, hipMemcpyDeviceToHost);
    printf("\n");

    for (int i = 0; i < M; ++i) {
        for(int j=0;j<4;j++){
            printf("%d,",static_cast<int>(int32b_arrayC[i*4+j]));
        }
        printf("\n");
    }
    printf("\n");
    printf("\n");
}



void gemv_perf_test(){
    int test_para[2048][5] = {
        {1024,1024},
        {2048,2048},
        {3072,3072},
        {4096,4096},
        {5120,5120},
        {6144,6144},
        {7168,7168},
        {8192,8192},
        {9216,9216},
        {10240,10240},
        {11264,11264},
        {12288,12288},
        {13312,13312},
        {14336,14336},
        {15360,15360},
        {16384,16384},
        {17408,17408},
        {18432,18432},
        {19456,19456},
        {20480,20480},
        {21504,21504},
        {22528,22528},
        {23552,23552},
        {24576,24576},
        {25600,25600},
        {26624,26624},
        {27648,27648},
        {28672,28672},
        {29696,29696},
        {30720,30720},
        {31744,31744},
        {32768,32768},
        }; 
  
    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    int max = 4096*8;
    float *matrixA = (float *)malloc(sizeof(float) * max*max);
    float *matrixB = (float *)malloc(sizeof(float) * max*max);
    float *matrixC = (float *)malloc(sizeof(float) * max*max);

    char * work;
    hipMalloc((char **)&work, sizeof(float) * (max*max*6+max*5));

    std::cout<<"M\tN\tSGEMV\n";
    const int digit = 8;
    
    float *A_d, *B_d, *C_d;
    generate_matrix<float>(matrixA,max,max,'u');
    generate_matrix<float>(matrixB,max,max,'u');   
    hipMalloc((float **)&A_d, sizeof(float) * max*max);
    hipMalloc((float **)&B_d, sizeof(float) * max);
    hipMalloc((float **)&C_d, sizeof(float) * max);
    hipMemcpy(A_d, matrixA, sizeof(float) * max*max, hipMemcpyHostToDevice);
    hipMemcpy(B_d, matrixB, sizeof(float) * max, hipMemcpyHostToDevice);    
    for(int i=0;i<32;i++){

        int N=test_para[i][0],M=test_para[i][1];
        
        float alpha = 1.0, beta = 0.0;
        if(M==0) return;
        hipDeviceSynchronize();
        std::cout<<M<<"\t"<<N<<"\t";

        //计算float和int矩阵乘法得到结果矩阵

        // xgemm(matrixA,matrixB,matrixC,M,K,K,N);
          beta = 0.0, alpha = 1.0;
        {
            auto start = std::chrono::high_resolution_clock::now();
            cublas_gemv_rowmajor( &cublasH,A_d, B_d, C_d, M, N, alpha, beta);
            hipDeviceSynchronize();
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double, std::milli> diff = end - start;
            double time  = diff.count();
            printf("%.7lf\n",time);
            hipDeviceSynchronize();
        }

    }
    return;        
}


void hgemm_perf_test(){

    int test_para[2048][3] = {
        {1024,1024,1024},
        {2048,2048,2048},
        {3072,3072,3072},
        {4096,4096,4096},
        {5120,5120,5120},
        {6144,6144,6144},
        {7168,7168,7168},
        {8192,8192,8192},
        {9216,9216,9216},
        {10240,10240,10240},
        {11264,11264,11264},
        {12288,12288,12288},
        {13312,13312,13312},
        {14336,14336,14336},
        {15360,15360,15360},
        {16384,16384,16384},
        {17408,17408,17408},
        {18432,18432,18432},
        {19456,19456,19456},
        {20480,20480,20480},
        {21504,21504,21504},
        {22528,22528,22528},
        {23552,23552,23552},
        {24576,24576,24576},
        {25600,25600,25600},
        {26624,26624,26624},
        {27648,27648,27648},
        {28672,28672,28672},
        {29696,29696,29696},
        {30720,30720,30720},
        {31744,31744,31744},
        {32768,32768,32768},
    }; 

    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    // 定义数组的大小

    int max = 4096*8;
    // 创建一个使用float类型的数组
    std::vector<half> int4b_arrayA(max*max);
    std::vector<half> int4b_arrayB(max*max);
    std::vector<half> int32b_arrayC(max*max);

    half* d_A;
    half* d_B;
    half* d_C;
    float* d_C32;
    hipMalloc((void**)&d_A, sizeof(half) * max*max);
    hipMalloc((void**)&d_B, sizeof(half) * max*max);
    hipMalloc((void**)&d_C, sizeof(half) * max*max);
    hipMalloc((void**)&d_C32, sizeof(float) * max*max);
    hipMemcpy(d_A, int4b_arrayA.data(), sizeof(half) * max*max, hipMemcpyHostToDevice);
    hipMemcpy(d_B, int4b_arrayB.data(), sizeof(half) * max*max, hipMemcpyHostToDevice);

    float beta = 0.0, alpha = 1.0;

    std::cout<<"M\tN\tK\th16-16_16\th16-16_32\th16-32_32\n";

    for(int i=0;i<32;i++){
        

        int N=test_para[i][0],M=test_para[i][1],K=test_para[i][2];
        half alpha = 1.0, beta = 0.0;
        if(M==0) return;

        std::cout<<M<<"\t"<<N<<"\t"<<K<<"\t";

        //计算float和int矩阵乘法得到结果矩阵
        cublas_gemm_rowmajor(
            &cublasH, d_A, d_B, d_C, M, K,
            K, N, alpha, beta);
        {
            auto start = std::chrono::high_resolution_clock::now();
            cublas_gemm_rowmajor(
                &cublasH, d_A, d_B, d_C, M, K,
                K, N, alpha, beta);
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double, std::milli> diff = end - start;
            double time  = diff.count();
            printf("%.7lf\t",time);
            hipDeviceSynchronize();
        }
        float alpha2 = 1.0, beta2 = 0.0;
        {
            auto start = std::chrono::high_resolution_clock::now();
            cublas_gemm_rowmajor(
                &cublasH, d_A, d_B, d_C, M, K,
                K, N, alpha2, beta2);
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double, std::milli> diff = end - start;
            double time  = diff.count();
            printf("%.7lf\t",time);
            hipDeviceSynchronize();
        }
        {
            auto start = std::chrono::high_resolution_clock::now();
            cublas_gemm_rowmajor(
                &cublasH, d_A, d_B, d_C32, M, K,
                K, N, alpha2, beta2);
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double, std::milli> diff = end - start;
            double time  = diff.count();
            printf("%.7lf\n",time);
            hipDeviceSynchronize();
        }


    }
    return;        

}

void mslag2d(float *in, half *out,int size){
    #pragma omp parallel for num_threads(max_omp_thread)
    for(int i=0; i<size; i++){
        out[i] = (half)in[i];
    }
}
void mdlag2s(half *in, float *out,int size){
    #pragma omp parallel for num_threads(max_omp_thread)
    for(int i=0; i<size; i++){
        out[i] = (float)in[i];
    }
}

void mslag2h_withR(float *in, half *out, float *res, float *pin,int size){
    #pragma omp parallel for num_threads(max_omp_thread)
    for(int i=0; i<size; i++){
        out[i] = (half)in[i];
        res[i] = in[i] - (float)out[i];
        pin[i] = (float)out[i];
    }
}


void hgemm1616_acc_test(){
    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    // 定义数组的大小
    int M=1024,N=1024,K=1024;
    // 创建一个使用float类型的数组
    std::vector<float> arrayA(M*K);
    std::vector<float> arrayB(K*N);
    std::vector<float> arrayC(M*N);
    std::vector<float> arrayhfC(M*N);


    generate_matrix<float>(arrayA.data(),M,K,'k');
    generate_matrix<float>(arrayB.data(),K,N,'k');    


    float* d_A;
    float* d_B;
    float* d_C;
    float* d_C_TMP;
    hipMalloc((void**)&d_A, sizeof(float) * M*K);
    hipMalloc((void**)&d_B, sizeof(float) * K*N);
    hipMalloc((void**)&d_C, sizeof(float) * M*N);
    hipMalloc((void**)&d_C_TMP, sizeof(float) * M*N);
    hipMemcpy(d_A, arrayA.data(), sizeof(float) * M*K, hipMemcpyHostToDevice);
    hipMemcpy(d_B, arrayB.data(), sizeof(float) * K*N, hipMemcpyHostToDevice);

    float  beta = 0.0, alpha = 1.0;




    cublas_gemm_rowmajor(
        &cublasH, d_A, d_B, d_C, M, K,
        K, N, alpha, beta);

    hipMemcpy( arrayC.data(),d_C, sizeof(float) * M*N, hipMemcpyDeviceToHost);
  

    // 创建一个使用half类型的数组
    std::vector<half> arrayhA(M*K);
    std::vector<half> arrayhB(K*N);
    std::vector<half> arrayhC(M*N);

    mslag2d(arrayA.data(),arrayhA.data(),M*K);
    mslag2d(arrayB.data(),arrayhB.data(),N*K);
    // mslag2d(arrayC.data(),arrayhC.data(),M*N);


    half* d_hA;
    half* d_hB;
    half* d_hC;
    hipMalloc((void**)&d_hA, sizeof(half) * M*K);
    hipMalloc((void**)&d_hB, sizeof(half) * K*N);
    hipMalloc((void**)&d_hC, sizeof(half) * M*N);
    hipMemcpy(d_hA, arrayhA.data(), sizeof(half) * M*K, hipMemcpyHostToDevice);
    hipMemcpy(d_hB, arrayhB.data(), sizeof(half) * K*N, hipMemcpyHostToDevice);
    

    half  beta2 = 0.0, alpha2 = 1.0;    
    cublas_gemm_rowmajor(
        &cublasH, d_hA, d_hB, d_hC, M, K,
        K, N, alpha2, beta2);
    hipDeviceSynchronize();

    hipMemcpy( arrayhC.data(),d_hC, sizeof(half) * M*N, hipMemcpyDeviceToHost);
  
    // for (int i = 0; i < M; ++i) {
    //     for(int j=0;j<N;j++){
    //         printf("%f,",(float)(arrayhB[i*N+j]));
    //     }
    //     printf("\n");
    // }

    mdlag2s(arrayhC.data(),arrayhfC.data(),M*N);
    float R3 = get_Ferror<float>(arrayC.data(),arrayhfC.data(),M,N); 

    printf("%.7f\n",R3);
}
void hgemm1632_acc_test(){
    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    // 定义数组的大小
    int M=1024,N=1024,K=1024;
    // 创建一个使用float类型的数组
    std::vector<float> arrayA(M*K);
    std::vector<float> arrayB(K*N);
    std::vector<float> arrayC(M*N);
    std::vector<float> arrayhfC(M*N);


    generate_matrix<float>(arrayA.data(),M,K,'k');
    generate_matrix<float>(arrayB.data(),K,N,'k');    


    float* d_A;
    float* d_B;
    float* d_C;
    float* d_C_TMP;
    hipMalloc((void**)&d_A, sizeof(float) * M*K);
    hipMalloc((void**)&d_B, sizeof(float) * K*N);
    hipMalloc((void**)&d_C, sizeof(float) * M*N);
    hipMalloc((void**)&d_C_TMP, sizeof(float) * M*N);
    hipMemcpy(d_A, arrayA.data(), sizeof(float) * M*K, hipMemcpyHostToDevice);
    hipMemcpy(d_B, arrayB.data(), sizeof(float) * K*N, hipMemcpyHostToDevice);

    float  beta = 0.0, alpha = 1.0;




    cublas_gemm_rowmajor(
        &cublasH, d_A, d_B, d_C, M, K,
        K, N, alpha, beta);

    hipMemcpy( arrayC.data(),d_C, sizeof(float) * M*N, hipMemcpyDeviceToHost);
  

    // 创建一个使用half类型的数组
    std::vector<half> arrayhA(M*K);
    std::vector<half> arrayhB(K*N);
    std::vector<float> arrayhC(M*N);

    mslag2d(arrayA.data(),arrayhA.data(),M*K);
    mslag2d(arrayB.data(),arrayhB.data(),N*K);
    // mslag2d(arrayC.data(),arrayhC.data(),M*N);


    half* d_hA;
    half* d_hB;
    float* d_hC;
    hipMalloc((void**)&d_hA, sizeof(half) * M*K);
    hipMalloc((void**)&d_hB, sizeof(half) * K*N);
    hipMalloc((void**)&d_hC, sizeof(float) * M*N);
    hipMemcpy(d_hA, arrayhA.data(), sizeof(half) * M*K, hipMemcpyHostToDevice);
    hipMemcpy(d_hB, arrayhB.data(), sizeof(half) * K*N, hipMemcpyHostToDevice);
    

    float  beta2 = 0.0, alpha2 = 1.0;    
    cublas_gemm_rowmajor(
        &cublasH, d_hA, d_hB, d_hC, M, K,
        K, N, alpha2, beta2);
    hipDeviceSynchronize();

    hipMemcpy( arrayhfC.data(),d_hC, sizeof(float) * M*N, hipMemcpyDeviceToHost);
  
    // for (int i = 0; i < M; ++i) {
    //     for(int j=0;j<N;j++){
    //         printf("%f,",(float)(arrayhB[i*N+j]));
    //     }
    //     printf("\n");
    // }
    float R3 = get_Ferror<float>(arrayC.data(),arrayhfC.data(),M,N); 

    printf("%.7f\n",R3);
}

int main(){
    //axpy_perf_test();
    //gemv_acc_test();
    //gemm_acc_test();
    //gemm_acc_test2();
    //hgemm_acc_test();

    // gemv_perf_test();
    // hgemm_perf_test();
    hgemm_perf_test();

    hgemm1616_acc_test();
    hgemm1632_acc_test();
    return 0;
}