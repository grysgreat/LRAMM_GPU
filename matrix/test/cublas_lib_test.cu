#include "../cublas/cublas_lib.cuh"
#include "../operator_matrix.cuh"
#include "../gen_matrix.cuh"
#include "../print_matrix.cuh"

#include <iomanip>
#include "stdio.h"
#include <chrono>




int axpy_perf_test(){
    int num = 8192;
    int N=num,M=num,K=num;

    float *matrixA = (float *)malloc(sizeof(float) * M*N);
    int8_t *matrixA8 = (int8_t *)malloc(sizeof(int8_t) * M*N);
    float *matrixB = (float *)malloc(sizeof(float) * M*N);

    float *vec_row = (float *)malloc(sizeof(float) * M*N);
    float *vec_col = (float *)malloc(sizeof(float) * M*N);
    float *work = (float *)malloc(sizeof(float) * M*N);
    generate_matrix<float>(matrixA,M,N,'u');
    generate_matrix<float>(matrixB,M,N,'u');

    float *matrixA_dev;
    float *matrixB_dev;
    float *work_dev;
    hipMalloc((void**)&matrixA_dev, sizeof(float) * M*N);
    hipMalloc((void**)&matrixB_dev, sizeof(float) * M*N);

    // start = std::chrono::high_resolution_clock::now();
    hipMemcpy(matrixA_dev, matrixA, sizeof(float) * M*N, hipMemcpyHostToDevice);
    hipMemcpy(matrixB_dev, matrixA, sizeof(float) * M*N, hipMemcpyHostToDevice);

    float alpha = 1.0;

    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;
    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));


    auto start = std::chrono::high_resolution_clock::now();
    cublas_saxpy(matrixA_dev,matrixB_dev,alpha, num*num,cublasH,stream);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = end - start;
    int time  = diff.count()*1000*1000;
    std::cout<<"size="<<M<<"//axpy - gpu time:" << std::fixed << std::setprecision(6) << time << std::endl;


    hipMemcpy(matrixB, matrixB_dev, sizeof(float) * M*N, hipMemcpyDeviceToHost);
}


void gemm_acc_test(){
    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    // 定义数组的大小
    int M=16,N=14,K=2;
    // 创建一个使用float类型的数组
    std::vector<float> int4b_arrayA(M*K);
    std::vector<float> int4b_arrayB(K*N);
    std::vector<float> int32b_arrayC(M*N);


    // 初始化数组
    for (int i = 0; i < M; ++i) {
        for(int j=0;j<K;j++){
            // 将每个元素初始化为它的索引值，注意这里只是示例，实际值可能需要根据量化规则来确定
            int4b_arrayA[i*K+j] = static_cast<float>(i*K+j);
        }
    }
    for (int i = 0; i < K; ++i) {
        for(int j=0;j<N;j++){
            // 将每个元素初始化为它的索引值，注意这里只是示例，实际值可能需要根据量化规则来确定
            int4b_arrayB[i*N+j] = static_cast<float>(j);
        }
    }


    for (int i = 0; i < M; ++i) {
        for(int j=0;j<K;j++){
            printf("%d,",static_cast<int>(int4b_arrayA[i*K+j]));
        }
        printf("\n");
    }
    printf("\n");
    for (int i = 0; i < K; ++i) {
        for(int j=0;j<N;j++){
            printf("%d,",static_cast<int>(int4b_arrayB[i*N+j]));
        }
        printf("\n");
    }
    printf("\n");

    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc((void**)&d_A, sizeof(float) * M*K);
    hipMalloc((void**)&d_B, sizeof(float) * K*N);
    hipMalloc((void**)&d_C, sizeof(float) * M*N);
    hipMemcpy(d_A, int4b_arrayA.data(), sizeof(float) * M*K, hipMemcpyHostToDevice);
    hipMemcpy(d_B, int4b_arrayB.data(), sizeof(float) * K*N, hipMemcpyHostToDevice);

    float  beta = 0.0, alpha = 1.0;
    // hipblasSgemm(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K,
    //             &alpha, d_A, M, d_B, M, &beta, d_C, M);  

    // hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K,
    //             &alpha, d_B, N, d_A, K, &beta, d_C, N);  

    cublas_gemm_rowmajor(
        &cublasH, d_A, d_B, d_C, M, K,
        K, N, alpha, beta);

    hipMemcpy( int32b_arrayC.data(),d_C, sizeof(float) * M*N, hipMemcpyDeviceToHost);
  
    for (int i = 0; i < M; ++i) {
        for(int j=0;j<N;j++){
            printf("%d,",static_cast<int>(int32b_arrayC[i*N+j]));
        }
        printf("\n");
    }

}

void gemv_acc_test(){
    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    // 定义数组的大小
    int M=32,N=16;
    // 创建一个使用float类型的数组
    std::vector<float> int4b_arrayA(M*N);
    std::vector<float> int4b_arrayB(N);
    std::vector<float> int32b_arrayC(M);


    // 初始化数组
    for (int i = 0; i < M; ++i) {
        for(int j=0;j<N;j++){
            // 将每个元素初始化为它的索引值，注意这里只是示例，实际值可能需要根据量化规则来确定
            int4b_arrayA[i*N+j] = static_cast<float>(i+1);
        }
    }
    for (int i = 0; i < N; ++i) int4b_arrayB[i] = i;
    for (int i = 0; i < M; ++i) int32b_arrayC[i] = 0;

    



    for (int i = 0; i < M; ++i) {
        for(int j=0;j<N;j++){
            printf("%d,",static_cast<int>(int4b_arrayA[i*N+j]));
        }
        printf("\n");
    }
    printf("\n");
    for (int i = 0; i < N; ++i) printf("%d,",static_cast<int>(int4b_arrayB[i]));

    printf("\n");

    float* d_A;
    float* d_B;
    float* d_C;
    float* d_A_tmp;
    hipMalloc((void**)&d_A, sizeof(float) * M*N);
    hipMalloc((void**)&d_A_tmp, sizeof(float) * M*N);
    hipMalloc((void**)&d_B, sizeof(float) * N);
    hipMalloc((void**)&d_C, sizeof(float) * M);
    hipMemcpy(d_A, int4b_arrayA.data(), sizeof(float) * M*N, hipMemcpyHostToDevice);
    hipMemcpy(d_B, int4b_arrayB.data(), sizeof(float) * N, hipMemcpyHostToDevice);

    float  beta = 0.0, alpha = 1.0;
    
    // strans(d_A_tmp,d_A,M,N);
    // hipblasSgemv(cublasH, HIPBLAS_OP_N, M, N, 
    //             &alpha, d_A_tmp, M, d_B, 1, &beta, d_C, 1);  

    hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, M, N,
                    &alpha, d_B, 1, d_A, N, &beta, d_C, 1);  
    hipDeviceSynchronize();

    hipMemcpy( int32b_arrayC.data(),d_C, sizeof(float) * M, hipMemcpyDeviceToHost);
    printf("\n");

    for (int i = 0; i < M; ++i) {
        printf("%d,",static_cast<int>(int32b_arrayC[i]));
    }
    printf("\n");
    printf("\n");
}


int main(){
    //axpy_perf_test();
    //gemv_acc_test();
      gemm_acc_test();
    return 0;
}