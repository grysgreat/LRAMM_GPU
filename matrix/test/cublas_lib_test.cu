#include "hip/hip_runtime.h"
#include "../cublas/cublas_lib.cuh"
#include "../operator_matrix.cuh"
#include "../gen_matrix.cuh"

#include <iomanip>
#include "stdio.h"
#include <chrono>


template <typename T>
T get_Ferror(T matrix_ref[],T matrix_cmp[],int rows,int cols){

    T sumR=0,sum=0;
    for(int i=0;i<rows;i++){
        for(int j=0;j<cols;j++){
            sumR+=(matrix_ref[i*cols+j] - matrix_cmp[i*cols+j])*(matrix_ref[i*cols+j] - matrix_cmp[i*cols+j]);
            sum+=(matrix_ref[i*cols+j])*(matrix_ref[i*cols+j]);
        }
    }

    T ans = sqrt(sumR)/sqrt(sum);
    return ans;

}


int axpy_perf_test(){
    int num = 8192;
    int N=num,M=num,K=num;

    float *matrixA = (float *)malloc(sizeof(float) * M*N);
    int8_t *matrixA8 = (int8_t *)malloc(sizeof(int8_t) * M*N);
    float *matrixB = (float *)malloc(sizeof(float) * M*N);

    float *vec_row = (float *)malloc(sizeof(float) * M*N);
    float *vec_col = (float *)malloc(sizeof(float) * M*N);
    float *work = (float *)malloc(sizeof(float) * M*N);
    generate_matrix<float>(matrixA,M,N,'u');
    generate_matrix<float>(matrixB,M,N,'u');

    float *matrixA_dev;
    float *matrixB_dev;
    float *work_dev;
    hipMalloc((void**)&matrixA_dev, sizeof(float) * M*N);
    hipMalloc((void**)&matrixB_dev, sizeof(float) * M*N);

    // start = std::chrono::high_resolution_clock::now();
    hipMemcpy(matrixA_dev, matrixA, sizeof(float) * M*N, hipMemcpyHostToDevice);
    hipMemcpy(matrixB_dev, matrixA, sizeof(float) * M*N, hipMemcpyHostToDevice);

    float alpha = 1.0;

    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;
    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));


    auto start = std::chrono::high_resolution_clock::now();
    cublas_saxpy(matrixA_dev,matrixB_dev,alpha, num*num,cublasH,stream);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = end - start;
    int time  = diff.count()*1000*1000;
    std::cout<<"size="<<M<<"//axpy - gpu time:" << std::fixed << std::setprecision(6) << time << std::endl;


    hipMemcpy(matrixB, matrixB_dev, sizeof(float) * M*N, hipMemcpyDeviceToHost);
}


void gemm_acc_test(){
    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    // 定义数组的大小
    int M=16,N=14,K=2;
    // 创建一个使用float类型的数组
    std::vector<float> int4b_arrayA(M*K);
    std::vector<float> int4b_arrayB(K*N);
    std::vector<float> int32b_arrayC(M*N);


    // 初始化数组
    for (int i = 0; i < M; ++i) {
        for(int j=0;j<K;j++){
            // 将每个元素初始化为它的索引值，注意这里只是示例，实际值可能需要根据量化规则来确定
            int4b_arrayA[i*K+j] = static_cast<float>(i*K+j);
        }
    }
    for (int i = 0; i < K; ++i) {
        for(int j=0;j<N;j++){
            // 将每个元素初始化为它的索引值，注意这里只是示例，实际值可能需要根据量化规则来确定
            int4b_arrayB[i*N+j] = static_cast<float>(j);
        }
    }


    for (int i = 0; i < M; ++i) {
        for(int j=0;j<K;j++){
            printf("%d,",static_cast<int>(int4b_arrayA[i*K+j]));
        }
        printf("\n");
    }
    printf("\n");
    for (int i = 0; i < K; ++i) {
        for(int j=0;j<N;j++){
            printf("%d,",static_cast<int>(int4b_arrayB[i*N+j]));
        }
        printf("\n");
    }
    printf("\n");

    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc((void**)&d_A, sizeof(float) * M*K);
    hipMalloc((void**)&d_B, sizeof(float) * K*N);
    hipMalloc((void**)&d_C, sizeof(float) * M*N);
    hipMemcpy(d_A, int4b_arrayA.data(), sizeof(float) * M*K, hipMemcpyHostToDevice);
    hipMemcpy(d_B, int4b_arrayB.data(), sizeof(float) * K*N, hipMemcpyHostToDevice);

    float  beta = 0.0, alpha = 1.0;
    // hipblasSgemm(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K,
    //             &alpha, d_A, M, d_B, M, &beta, d_C, M);  

    // hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K,
    //             &alpha, d_B, N, d_A, K, &beta, d_C, N);  

    cublas_gemm_rowmajor(
        &cublasH, d_A, d_B, d_C, M, K,
        K, N, alpha, beta);

    hipMemcpy( int32b_arrayC.data(),d_C, sizeof(float) * M*N, hipMemcpyDeviceToHost);
  
    for (int i = 0; i < M; ++i) {
        for(int j=0;j<N;j++){
            printf("%d,",static_cast<int>(int32b_arrayC[i*N+j]));
        }
        printf("\n");
    }

}


void gemm_acc_test2(){
    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    // 定义数组的大小
    int M=16,N=14,K=2;
    // 创建一个使用float类型的数组
    std::vector<float> int4b_arrayA(M*K);
    std::vector<float> int4b_arrayB(K*N);
    std::vector<float> int32b_arrayC(M*N);


    // 初始化数组
    for (int i = 0; i < M; ++i) {
        for(int j=0;j<K;j++){
            // 将每个元素初始化为它的索引值，注意这里只是示例，实际值可能需要根据量化规则来确定
            int4b_arrayA[i*K+j] = static_cast<float>(i*K+j);
        }
    }
    for (int i = 0; i < K; ++i) {
        for(int j=0;j<N;j++){
            // 将每个元素初始化为它的索引值，注意这里只是示例，实际值可能需要根据量化规则来确定
            int4b_arrayB[i*N+j] = static_cast<float>(j);
        }
    }


    for (int i = 0; i < M; ++i) {
        for(int j=0;j<K;j++){
            printf("%d,",static_cast<int>(int4b_arrayA[i*K+j]));
        }
        printf("\n");
    }
    printf("\n");
    for (int i = 0; i < K; ++i) {
        for(int j=0;j<N;j++){
            printf("%d,",static_cast<int>(int4b_arrayB[i*N+j]));
        }
        printf("\n");
    }
    printf("\n");

    float* d_A;
    float* d_B;
    float* d_C;
    float* d_C_TMP;
    hipMalloc((void**)&d_A, sizeof(float) * M*K);
    hipMalloc((void**)&d_B, sizeof(float) * K*N);
    hipMalloc((void**)&d_C, sizeof(float) * M*N);
    hipMalloc((void**)&d_C_TMP, sizeof(float) * M*N);
    hipMemcpy(d_A, int4b_arrayA.data(), sizeof(float) * M*K, hipMemcpyHostToDevice);
    hipMemcpy(d_B, int4b_arrayB.data(), sizeof(float) * K*N, hipMemcpyHostToDevice);

    float  beta = 0.0, alpha = 1.0;
    // hipblasSgemm(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K,
    //             &alpha, d_A, M, d_B, M, &beta, d_C, M);  

    // hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K,
    //             &alpha, d_B, N, d_A, K, &beta, d_C, N);  


    hipblasSgemm(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, M, N, K, &alpha, d_A, K, d_B, K, &beta, d_C_TMP, M);
    hipblasSgeam(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, N, M, &alpha, d_C_TMP, M, &beta, d_C, N, d_C, N);
    hipDeviceSynchronize();


    hipMemcpy( int32b_arrayC.data(),d_C, sizeof(float) * M*N, hipMemcpyDeviceToHost);
  
    for (int i = 0; i < M; ++i) {
        for(int j=0;j<N;j++){
            printf("%d,",static_cast<int>(int32b_arrayC[i*N+j]));
        }
        printf("\n");
    }

}


void gemv_acc_test(){
    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    // 定义数组的大小
    int M=32,N=16;
    // 创建一个使用float类型的数组
    std::vector<float> int4b_arrayA(M*N);
    std::vector<float> int4b_arrayB(N);
    std::vector<float> int32b_arrayC(M*4);


    // 初始化数组
    for (int i = 0; i < M; ++i) {
        for(int j=0;j<N;j++){
            // 将每个元素初始化为它的索引值，注意这里只是示例，实际值可能需要根据量化规则来确定
            int4b_arrayA[i*N+j] = static_cast<float>(i+1);
        }
    }
    for (int i = 0; i < N; ++i) int4b_arrayB[i] = i;
    for (int i = 0; i < M; ++i) int32b_arrayC[i] = 0;

    



    for (int i = 0; i < M; ++i) {
        for(int j=0;j<N;j++){
            printf("%d,",static_cast<int>(int4b_arrayA[i*N+j]));
        }
        printf("\n");
    }
    printf("\n");
    for (int i = 0; i < N; ++i) printf("%d,",static_cast<int>(int4b_arrayB[i]));

    printf("\n");

    float* d_A;
    float* d_B;
    float* d_C;
    float* d_A_tmp;
    hipMalloc((void**)&d_A, sizeof(float) * M*N);
    hipMalloc((void**)&d_A_tmp, sizeof(float) * M*N);
    hipMalloc((void**)&d_B, sizeof(float) * N);
    hipMalloc((void**)&d_C, sizeof(float) * M);
    hipMemcpy(d_A, int4b_arrayA.data(), sizeof(float) * M*N, hipMemcpyHostToDevice);
    hipMemcpy(d_B, int4b_arrayB.data(), sizeof(float) * N, hipMemcpyHostToDevice);

    float  beta = 0.0, alpha = 1.0;
    
    // strans(d_A_tmp,d_A,M,N);
    // hipblasSgemv(cublasH, HIPBLAS_OP_N, M, N, 
    //             &alpha, d_A_tmp, M, d_B, 1, &beta, d_C, 1);  

    hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, 4, 4,
                    &alpha, d_B, 1, d_A, N, &beta, d_C, 4);  
    hipDeviceSynchronize();

    hipMemcpy( int32b_arrayC.data(),d_C, sizeof(float) * M, hipMemcpyDeviceToHost);
    printf("\n");

    for (int i = 0; i < M; ++i) {
        for(int j=0;j<4;j++){
            printf("%d,",static_cast<int>(int32b_arrayC[i*4+j]));
        }
        printf("\n");
    }
    printf("\n");
    printf("\n");
}



void performance_test(){
    int test_para[2048][5] = {

        // {2048,2048,2048,50},
        // {2048,2048,2048,40},
        // {2048,2048,2048,30},
        // {2048,2048,2048,20},
        // {2048,2048,2048,10},
        // {2048,2048,2048,1},
        // {16384,16384,16384,1},
    {128,128},
    {256,256},
    {384,384},
    {512,512},
    {640,640},
    {768,768},
    {896,896},
    {1024,1024},
    {1152,1152},
    {1280,1280},
    {1408,1408},
    {1536,1536},
    {1664,1664},
    {1792,1792},
    {1920,1920},
    {2048,2048},
    {2176,2176},
    {2304,2304},
    {2432,2432},
    {2560,2560},
    {2688,2688},
    {2816,2816},
    {2944,2944},
    {3072,3072},
    {3200,3200},
    {3328,3328},
    {3456,3456},
    {3584,3584},
    {3712,3712},
    {3840,3840},
    {3968,3968},
    {4096,4096},
    {4224,4224},
    {4352,4352},
    {4480,4480},
    {4608,4608},
    {4736,4736},
    {4864,4864},
    {4992,4992},
    {5120,5120},
    {5248,5248},
    {5376,5376},
    {5504,5504},
    {5632,5632},
    {5760,5760},
    {5888,5888},
    {6016,6016},
    {6144,6144},
    {6272,6272},
    {6400,6400},
    {6528,6528},
    {6656,6656},
    {6784,6784},
    {6912,6912},
    {7040,7040},
    {7168,7168},
    {7296,7296},
    {7424,7424},
    {7552,7552},
    {7680,7680},
    {7808,7808},
    {7936,7936},
    {8064,8064},
    {8192,8192},
    {8320,8320},
    {8448,8448},
    {8576,8576},
    {8704,8704},
    {8832,8832},
    {8960,8960},
    {9088,9088},
    {9216,9216},
    {9344,9344},
    {9472,9472},
    {9600,9600},
    {9728,9728},
    {9856,9856},
    {9984,9984},
    {10112,10112},
    {10240,10240},
    {10368,10368},
    {10496,10496},
    {10624,10624},
    {10752,10752},
    {10880,10880},
    {11008,11008},
    {11136,11136},
    {11264,11264},
    {11392,11392},
    {11520,11520},
    {11648,11648},
    {11776,11776},
    {11904,11904},
    {12032,12032},
    {12160,12160},
    {12288,12288},
    {12416,12416},
    {12544,12544},
    {12672,12672},
    {12800,12800},
    {12928,12928},
    {13056,13056},
    {13184,13184},
    {13312,13312},
    {13440,13440},
    {13568,13568},
    {13696,13696},
    {13824,13824},
    {13952,13952},
    {14080,14080},
    {14208,14208},
    {14336,14336},
    {14464,14464},
    {14592,14592},
    {14720,14720},
    {14848,14848},
    {14976,14976},
    {15104,15104},
    {15232,15232},
    {15360,15360},
    {15488,15488},
    {15616,15616},
    {15744,15744},
    {15872,15872},
    {16000,16000},
    {16128,16128},
    {16256,16256},
        }; 
  
    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    int max = 4096*4;
    float *matrixA = (float *)malloc(sizeof(float) * max*max);
    float *matrixB = (float *)malloc(sizeof(float) * max*max);
    float *matrixC = (float *)malloc(sizeof(float) * max*max);

    char * work;
    hipMalloc((char **)&work, sizeof(float) * (max*max*6+max*5));

    std::cout<<"M\tN\tSGEMV\n";
    const int digit = 8;
    
    float *A_d, *B_d, *C_d;
    generate_matrix<float>(matrixA,max,max,'u');
    generate_matrix<float>(matrixB,max,max,'u');   
    hipMalloc((float **)&A_d, sizeof(float) * max*max);
    hipMalloc((float **)&B_d, sizeof(float) * max);
    hipMalloc((float **)&C_d, sizeof(float) * max);
    hipMemcpy(A_d, matrixA, sizeof(float) * max*max, hipMemcpyHostToDevice);
    hipMemcpy(B_d, matrixB, sizeof(float) * max, hipMemcpyHostToDevice);    
    for(int i=0;i<128;i++){

        int N=test_para[i][0],M=test_para[i][1];
        
        float alpha = 1.0, beta = 0.0;
        if(M==0) return;
        hipDeviceSynchronize();
        std::cout<<M<<"\t"<<N<<"\t";

        //计算float和int矩阵乘法得到结果矩阵

        // xgemm(matrixA,matrixB,matrixC,M,K,K,N);
          beta = 0.0, alpha = 1.0;
        {
            auto start = std::chrono::high_resolution_clock::now();
            cublas_gemv_rowmajor( &cublasH,A_d, B_d, C_d, M, N, alpha, beta);
            hipDeviceSynchronize();
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double, std::milli> diff = end - start;
            double time  = diff.count();
            printf("%.7lf\n",time);
            hipDeviceSynchronize();
        }

    }
    return;        
}


void hgemm_perf_test(){
    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    // 定义数组的大小
    int M=8192,N=8192,K=8192;
    // 创建一个使用float类型的数组
    std::vector<half> int4b_arrayA(M*K);
    std::vector<half> int4b_arrayB(K*N);
    std::vector<half> int32b_arrayC(M*N);

    half* d_A;
    half* d_B;
    half* d_C;
    hipMalloc((void**)&d_A, sizeof(half) * M*K);
    hipMalloc((void**)&d_B, sizeof(half) * K*N);
    hipMalloc((void**)&d_C, sizeof(half) * M*N);
    hipMemcpy(d_A, int4b_arrayA.data(), sizeof(half) * M*K, hipMemcpyHostToDevice);
    hipMemcpy(d_B, int4b_arrayB.data(), sizeof(half) * K*N, hipMemcpyHostToDevice);

    float beta = 0.0, alpha = 1.0;


    cublas_gemm_rowmajor(
        &cublasH, d_A, d_B, d_C, M, K,
        K, N, alpha, beta);

    auto start = std::chrono::high_resolution_clock::now();
    cublas_gemm_rowmajor(
        &cublasH, d_A, d_B, d_C, M, K,
        K, N, alpha, beta);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> diff = end - start;
    double time  = diff.count();
    std::cout<< std::fixed << std::setprecision(6) << time << "\n";
 



}

void mslag2d(float *in, half *out,int size){
    #pragma omp parallel for num_threads(max_omp_thread)
    for(int i=0; i<size; i++){
        out[i] = (half)in[i];
    }
}
void mdlag2s(half *in, float *out,int size){
    #pragma omp parallel for num_threads(max_omp_thread)
    for(int i=0; i<size; i++){
        out[i] = (float)in[i];
    }
}

void hgemm_acc_test(){
    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    // 定义数组的大小
    int M=1024,N=1024,K=1024;
    // 创建一个使用float类型的数组
    std::vector<float> arrayA(M*K);
    std::vector<float> arrayB(K*N);
    std::vector<float> arrayC(M*N);
    std::vector<float> arrayhfC(M*N);


    generate_matrix<float>(arrayA.data(),M,K,'k');
    generate_matrix<float>(arrayB.data(),K,N,'k');    


    float* d_A;
    float* d_B;
    float* d_C;
    float* d_C_TMP;
    hipMalloc((void**)&d_A, sizeof(float) * M*K);
    hipMalloc((void**)&d_B, sizeof(float) * K*N);
    hipMalloc((void**)&d_C, sizeof(float) * M*N);
    hipMalloc((void**)&d_C_TMP, sizeof(float) * M*N);
    hipMemcpy(d_A, arrayA.data(), sizeof(float) * M*K, hipMemcpyHostToDevice);
    hipMemcpy(d_B, arrayB.data(), sizeof(float) * K*N, hipMemcpyHostToDevice);

    float  beta = 0.0, alpha = 1.0;




    cublas_gemm_rowmajor(
        &cublasH, d_A, d_B, d_C, M, K,
        K, N, alpha, beta);

    hipMemcpy( arrayC.data(),d_C, sizeof(float) * M*N, hipMemcpyDeviceToHost);
  

    // 创建一个使用half类型的数组
    std::vector<half> arrayhA(M*K);
    std::vector<half> arrayhB(K*N);
    std::vector<half> arrayhC(M*N);

    mslag2d(arrayA.data(),arrayhA.data(),M*K);
    mslag2d(arrayB.data(),arrayhB.data(),N*K);
    // mslag2d(arrayC.data(),arrayhC.data(),M*N);


    half* d_hA;
    half* d_hB;
    half* d_hC;
    hipMalloc((void**)&d_hA, sizeof(half) * M*K);
    hipMalloc((void**)&d_hB, sizeof(half) * K*N);
    hipMalloc((void**)&d_hC, sizeof(half) * M*N);
    hipMemcpy(d_hA, arrayhA.data(), sizeof(half) * M*K, hipMemcpyHostToDevice);
    hipMemcpy(d_hB, arrayhB.data(), sizeof(half) * K*N, hipMemcpyHostToDevice);
    

    half  beta2 = 0.0, alpha2 = 1.0;    
    cublas_gemm_rowmajor(
        &cublasH, d_hA, d_hB, d_hC, M, K,
        K, N, alpha2, beta2);
    hipDeviceSynchronize();

    hipMemcpy( arrayhC.data(),d_hC, sizeof(half) * M*N, hipMemcpyDeviceToHost);
  
    // for (int i = 0; i < M; ++i) {
    //     for(int j=0;j<N;j++){
    //         printf("%f,",(float)(arrayhB[i*N+j]));
    //     }
    //     printf("\n");
    // }

    mdlag2s(arrayhC.data(),arrayhfC.data(),M*N);
    float R3 = get_Ferror<float>(arrayC.data(),arrayhfC.data(),M,N); 

    printf("%.7f\n",R3);
}


int main(){
    //axpy_perf_test();
    //gemv_acc_test();
    //gemm_acc_test();
    //gemm_acc_test2();
    //hgemm_acc_test();
    performance_test();
    return 0;
}