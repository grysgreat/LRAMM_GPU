#include "hip/hip_runtime.h"
#include <random>
#include <chrono>
#include <iomanip>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <omp.h>
#include "../cusolver_connector.cuh"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "../operator_matrix.cuh"
#include <chrono>
#include "./cublas/cublas_lib.cuh"
template <typename T>
void generate_matrix(std::vector<T> &matrix,int rows,int cols,char type ){
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(0.0, 1.0);
    if(type == 'u'){
        for(int i=0;i<rows;i++){
            for(int j=0;j<cols;j++){
                matrix[i*cols+j] = dis(gen);
                if(i==j&&i!=rows-1) matrix[i*cols+j] = i*cols+j;// (matrix[i*cols+j]);
                else  matrix[i*cols+j]=i*cols+j;//(matrix[i*cols+j]);
            }
        }     
    }
}
void print_Matrix(float matrix[], int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            int index = i * cols + j;
            printf("%.1f ", matrix[index]);
        }
        std::cout << std::endl;
    }
}


int cus_test() {



    using data_type = float;
    /* Input matrix dimensions */
    const int64_t m = 16;
    const int64_t n = 16;
    const int64_t lda = m;;
    const int64_t ldu = m;
    const int64_t ldv = n;
    /* rank of matrix A */
    const int64_t min_mn = std::min(m, n);

    std::vector<data_type> A(m*n);

    generate_matrix<float>(A,m,n,'u');

    int M=m,N=n;
    print_Matrix(A.data(),M,N);

    data_type *d_A = nullptr;
    data_type *d_U = nullptr;
    data_type *d_S = nullptr;
    data_type *d_V = nullptr;
    data_type *d_AO = nullptr;

    data_type *U = (data_type *)malloc(sizeof(data_type) * m*n);
    data_type *V = (data_type *)malloc(sizeof(data_type) * m*n);
    data_type *S = (data_type *)malloc(sizeof(data_type) * m*n);

    int rank = 2;
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_AO), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_U), sizeof(data_type) * ldu * m));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_V), sizeof(data_type) * ldv * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_S), sizeof(data_type) * min_mn));

    CUDA_CHECK(hipMemcpy(d_A, A.data(), sizeof(data_type) * lda * n, hipMemcpyHostToDevice));


    hipsolverHandle_t cusolverH = NULL;

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    cusolver_rsvd(m, n, d_A, d_U,d_V,d_S, rank, &cusolverH);

    hipMemcpy( U,d_U, sizeof(data_type) * M*N, hipMemcpyDeviceToHost);
    hipMemcpy( V,d_V, sizeof(data_type) * M*N, hipMemcpyDeviceToHost);
    hipMemcpy( S,d_S, sizeof(data_type) * min_mn, hipMemcpyDeviceToHost);


    printf("\n");
    print_Matrix(U,M,N);
    printf("\n");
    print_Matrix(V,M,N);
    printf("\n");
    print_Matrix(S,M,N);

    diag_matmul(d_V, d_S, rank, n);

    hipMemcpy( V,d_V, sizeof(data_type) * M*N, hipMemcpyDeviceToHost);
    printf("\n");
    print_Matrix(V,M,N);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float alpha = 1.0, beta = 0.0;



    strans(d_U,d_U,rank,m);
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, rank,
                &alpha, d_U, m, d_V, n, &beta, d_AO, n);    

    hipMemcpy( U,d_AO, sizeof(data_type) * M*N, hipMemcpyDeviceToHost);
    printf("\n");
    print_Matrix(U,M,N);

}

void lr_acc_test(){
    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    int M=8,N=8;
    int rank = 2;
    std::vector<float> int4b_arrayA(M*N);
    std::vector<float> int4b_arrayB(M*rank);
    std::vector<float> int32b_arrayC(N*rank);

    hipsolverHandle_t cusolverH = NULL;

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    for (int i = 0; i < M; ++i) {
        for(int j=0;j<N;j++){
            int4b_arrayA[i*N+j] = static_cast<float>(i+1);
        }
    }




    for (int i = 0; i < M; ++i) {
        for(int j=0;j<N;j++){
            printf("%d,",static_cast<int>(int4b_arrayA[i*N+j]));
        }
        printf("\n");
    }

    printf("\n");

    float* d_A;
    float* d_B;
    float* d_C;
    float* d_C_tmp;
    float* d_B_tmp;
    float* d_A_tmp;
    hipMalloc((void**)&d_A, sizeof(float) * M*N);
    hipMalloc((void**)&d_A_tmp, sizeof(float) * M*N);
    hipMalloc((void**)&d_B, sizeof(float) * M*rank);
    hipMalloc((void**)&d_B_tmp, sizeof(float) * M*rank);
    hipMalloc((void**)&d_C, sizeof(float) * N*rank);
    hipMalloc((void**)&d_C_tmp, sizeof(float) * N*rank);
    hipMemcpy(d_A, int4b_arrayA.data(), sizeof(float) * M*N, hipMemcpyHostToDevice);



    strans(d_A_tmp,d_A,M,N);

    // cusolver_rsvd_LR2(N, M, d_A_tmp, d_C, d_B, rank, &cusolverH);
    cusolver_rsvd_LR2(M, N, d_A_tmp, d_B, d_C, rank, &cusolverH);
    // strans(d_C,d_C_tmp,rank,N);
    // strans(d_B,d_B_tmp,M,rank);

    hipDeviceSynchronize();


    hipMemcpy( int4b_arrayB.data(), d_B, sizeof(float) * M *rank, hipMemcpyDeviceToHost);
    hipMemcpy( int32b_arrayC.data(), d_C, sizeof(float) * N*rank, hipMemcpyDeviceToHost);
    printf("\n");

    for (int i = 0; i < M; ++i) {
        for(int j=0;j<rank;j++){
            printf("%f,",(int4b_arrayB[i*rank+j]));
        }
        printf("\n");    
    }
    printf("\n");
    for (int i = 0; i < rank; ++i) {
        for(int j=0;j<N;j++){
            printf("%f,",(int32b_arrayC[i*N+j]));
        }
        printf("\n");
    }
    printf("\n");
    printf("\n");

    // cublas_gemm_rowmajor(
    //     &cublasH, d_B, d_C, d_A,  M,  rank,
    //     rank,  N, 1.0,  0.0);    
    float alpha = 1.0, beta = 0.0;
    hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, rank,
                &alpha, d_B, M, d_C, rank, &beta, d_A, M);  
    hipDeviceSynchronize();

    hipMemcpy( int4b_arrayA.data(), d_A, sizeof(float) * N* M, hipMemcpyDeviceToHost);
    for (int i = 0; i < M; ++i) {
        for(int j=0;j<N;j++){
            printf("%d,",static_cast<int>(int4b_arrayA[i*N+j]));
        }
        printf("\n");
    }


}

int main(){
    // cus_test();
    lr_acc_test();
}