#include "hip/hip_runtime.h"
#include "../lrxigemm.cuh"
#include <chrono>
#include "../gen_matrix.cuh"


void print_MatrixE(float matrix[], int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            int index = i * cols + j;
            printf("%.2e ", matrix[index]);
        }
        std::cout << std::endl;
    }
}
template <typename T>
void xgemm(const T A[], const T B[], T C[], int rowsA, int colsA, int rowsB, int colsB) {


    T* tmp =(T *)malloc(sizeof(T) * rowsA*colsB);

    for (int i = 0; i < rowsA; ++i) {
        for (int j = 0; j < colsB; ++j) {
            tmp[i * colsB + j] = 0;
            for (int k = 0; k < colsA; ++k) {
                tmp[i * colsB + j] += A[i * colsA + k] * B[k * colsB + j];
            }
        }
    }
    
    for (int i = 0; i < rowsA; ++i) {
        for (int j = 0; j < colsB; ++j) {
            C[i * colsB + j]  = tmp[i * colsB + j];
		}
	}
    
}

template <typename T>
T get_Ferror(T matrix_ref[],T matrix_cmp[],int rows,int cols){

    T sumR=0,sum=0;
    for(int i=0;i<rows;i++){
        for(int j=0;j<cols;j++){
            sumR+=(matrix_ref[i*cols+j] - matrix_cmp[i*cols+j])*(matrix_ref[i*cols+j] - matrix_cmp[i*cols+j]);
            sum+=(matrix_ref[i*cols+j])*(matrix_ref[i*cols+j]);
        }
    }

    T ans = sqrt(sumR)/sqrt(sum);
    return ans;

}



void  curand_test(){
    int len = 32;
    float *Sketch;
    hiprandGenerator_t gen;
    hiprandGenerator_t *gen_p=&gen;
    hipMalloc((void**)&Sketch, sizeof(float) * len);
    hiprandCreateGenerator(gen_p, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandGenerateNormal(*gen_p, Sketch, len, 0.0f, 1.0f);

    float *Sketch_h = (float *)malloc(sizeof(float) * len);

    hipMemcpy(Sketch_h, Sketch, sizeof(float) * len, hipMemcpyDeviceToHost);
    for(int j=0;j<len;j++){
        printf("%.4f, ",Sketch_h[j]);
    }    
    printf("\n\n");
    return;

}

void sketch_acc_test(){
    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    int M=32,N=16;

    std::vector<float> int4b_arrayA(M*N);
    std::vector<float> int4b_arrayB(M);
    std::vector<float> int32b_arrayC(N);


    for (int i = 0; i < M; ++i) {
        for(int j=0;j<N;j++){
            int4b_arrayA[i*N+j] = static_cast<float>(i+1);
        }
    }




    for (int i = 0; i < M; ++i) {
        for(int j=0;j<N;j++){
            printf("%d,",static_cast<int>(int4b_arrayA[i*N+j]));
        }
        printf("\n");
    }

    printf("\n");

    float* d_A;
    float* d_B;
    float* d_C;
    float* d_A_tmp;
    hipMalloc((void**)&d_A, sizeof(float) * M*N);
    hipMalloc((void**)&d_A_tmp, sizeof(float) * M*N);
    hipMalloc((void**)&d_B, sizeof(float) * M);
    hipMalloc((void**)&d_C, sizeof(float) * N);
    hipMemcpy(d_A, int4b_arrayA.data(), sizeof(float) * M*N, hipMemcpyHostToDevice);

    hiprandGenerator_t gen;

    /*prepare work space*/
    int threadsPerBlock = 2048; 
    int max_work_size = (max(M*N, 1)+threadsPerBlock-1)/threadsPerBlock;

    float* c_work = (float *)malloc(sizeof(float) * max_work_size);
    float* d_work;
    hipMalloc((float **)&d_work, sizeof(float) * max_work_size);

    sketch_r1_re(
        d_A,  d_B, d_C, M, N, &gen, &cublasH
    );
    


    hipMemcpy( int4b_arrayB.data(), d_B, sizeof(float) * M, hipMemcpyDeviceToHost);
    hipMemcpy( int32b_arrayC.data(), d_C, sizeof(float) * N, hipMemcpyDeviceToHost);
    printf("\n");

    for (int i = 0; i < M; ++i) {
        printf("%f,",(int4b_arrayB[i]));
    }
    printf("\n");
    for (int i = 0; i < N; ++i) {
        printf("%f,",(int32b_arrayC[i]));
    }
    printf("\n");
    printf("\n");

    cublas_gemm_rowmajor(
        &cublasH, d_B, d_C, d_A,  M,  1,
        1,  N, 1.0,  0.0);    
    hipMemcpy( int4b_arrayA.data(), d_A, sizeof(float) * N* M, hipMemcpyDeviceToHost);
    for (int i = 0; i < M; ++i) {
        for(int j=0;j<N;j++){
            printf("%d,",static_cast<int>(int4b_arrayA[i*N+j]));
        }
        printf("\n");
    }


}




void skxigemm_acc(){
    int max = 1024;
    float *matrixA = (float *)malloc(sizeof(float) * max*max);
    float *matrixB = (float *)malloc(sizeof(float) * max*max);
    float *matrixC = (float *)malloc(sizeof(float) * max*max);
    float *matrixCQ = (float *)malloc(sizeof(float) * max*max);
    float *matrixR = (float *)malloc(sizeof(float) * max*max);

    int M=max , N=max, K = max;

    generate_matrix<float>(matrixA,M,K,'u');
    generate_matrix<float>(matrixB,K,N,'u');    

    xgemm(matrixA,matrixB,matrixC,M,K,K,N);

    float *A_d, *B_d, *C_d;
    hipMalloc((float **)&A_d, sizeof(float) * M*K);
    hipMalloc((float **)&B_d, sizeof(float) * K*N);
    hipMalloc((float **)&C_d, sizeof(float) * M*N);
    hipMemcpy(A_d, matrixA, sizeof(float) * M*K, hipMemcpyHostToDevice);
    hipMemcpy(B_d, matrixB, sizeof(float) * K*N, hipMemcpyHostToDevice);

    xigemm<float,8>(A_d,B_d,C_d,M,K,K,N);
    hipMemcpy( matrixCQ,C_d, sizeof(float) * M * N, hipMemcpyDeviceToHost);
    float R2 = get_Ferror<float>(matrixC,matrixCQ,M,N); 
    printf("%.7f\n",R2);

    hipblasHandle_t cublasH = NULL;
    hipsolverHandle_t cusolverH = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    skxigemm<float,8>(A_d,B_d,C_d,M,K,K,N,10, &cusolverH, &cublasH);
    hipMemcpy( matrixCQ,C_d, sizeof(float) * M * N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    float R3 = get_Ferror<float>(matrixC,matrixCQ,M,N); 

    printf("%.7f\n",R3);


    return ;
}



void precision_test(){
    int test_para[2048][5] = {

        // {2048,2048,2048,50},
        // {2048,2048,2048,40},
        // {2048,2048,2048,30},
        // {2048,2048,2048,20},
        // {2048,2048,2048,10},
        {256,256,256,10,'n'},
        {4096,4096,4096,10,'n'},
        {4096,4096,4096,10,'u'},
        {4096,4096,4096,10,'s'},
        {4096,4096,4096,10,'e'},
        {4096,4096,4096,10,'k'},
        {4096,4096,4096,10,'p'},

        // {128,128,128,10,'k'},
        // {256,256,256,10,'k'},
        // {384,384,384,10,'k'},
        // {512,512,512,10,'k'},
        // {640,640,640,10,'k'},
        // {768,768,768,10,'k'},
        // {896,896,896,10,'k'},
        // {2048,2048,2048,10,'k'},
        // {1152,1152,1152,10,'k'},
        // {1280,1280,1280,10,'k'},
        // {1408,1408,1408,10,'k'},
        // {1536,1536,1536,10,'k'},
        // {1664,1664,1664,10,'k'},
        // {1792,1792,1792,10,'k'},
        // {1920,1920,1920,10,'k'},
        // {2048,2048,2048,10,'k'},
        // {2176,2176,2176,10,'k'},
        // {2304,2304,2304,10,'k'},
        // {2432,2432,2432,10,'k'},
        // {2560,2560,2560,10,'k'},
        // {2688,2688,2688,10,'k'},
        // {2816,2816,2816,10,'k'},
        // {2944,2944,2944,10,'k'},
        // {3072,3072,3072,10,'k'},
        // {3200,3200,3200,10,'k'},
        // {3328,3328,3328,10,'k'},
    }; 
  
    hipblasHandle_t cublasH = NULL;
    hipsolverHandle_t cusolverH = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    int max = 8192;
    float *matrixA = (float *)malloc(sizeof(float) * max*max);
    float *matrixB = (float *)malloc(sizeof(float) * max*max);
    float *matrixC = (float *)malloc(sizeof(float) * max*max);
    float *matrixCQ = (float *)malloc(sizeof(float) * max*max);
    float *matrixR = (float *)malloc(sizeof(float) * max*max);

    std::cout<<"M\tN\tK\ttype\trank\torigin\t\tLrxigemm\tsketch\n";
    const int digit = 8;
    char * work;
    hipMalloc((char **)&work, sizeof(float) * (max*max*8+max*5));

    float *A_d, *B_d, *C_d;
    for(int i=0;i<5;i++){
        

        int N=test_para[i][0],M=test_para[i][1],K=test_para[i][2];
        int rank =test_para[i][3];
        char type =test_para[i][4];

        float alpha = 1.0, beta = 0.0;

        if(i!=0) {
            if(N!=test_para[i-1][0]||M!=test_para[i-1][1]||K!=test_para[i-1][2]||type!=test_para[i-1][4]){
                generate_matrix<float>(matrixA,M,K,type);
                generate_matrix<float>(matrixB,K,N,type);
            }
        } else {
            generate_matrix<float>(matrixA,M,K,type);
            generate_matrix<float>(matrixB,K,N,type);            
        }
        hipMalloc((float **)&A_d, sizeof(float) * M*K);
        hipMalloc((float **)&B_d, sizeof(float) * K*N);
        hipMalloc((float **)&C_d, sizeof(float) * M*N);
        hipMemcpy(A_d, matrixA, sizeof(float) * M*K, hipMemcpyHostToDevice);
        hipMemcpy(B_d, matrixB, sizeof(float) * K*N, hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        std::cout<<M<<"\t"<<N<<"\t"<<K<<"\t"<<type<<"\t"<<rank<<"\t";

        //计算float和int矩阵乘法得到结果矩阵

        // xgemm(matrixA,matrixB,matrixC,M,K,K,N);
        cublas_gemm_rowmajor(
            &cublasH, A_d, B_d, C_d, M, K,
            K, N, alpha, beta);
        hipMemcpy( matrixC,C_d, sizeof(float) * M * N, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();




        {
            xigemm_mem<float,8>(A_d,B_d,C_d,work,M,K,K,N);
            hipMemcpy( matrixCQ,C_d, sizeof(float) * M * N, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            float R2 = get_Ferror<float>(matrixC,matrixCQ,M,N); 
            printf("%.7f\t",R2);
            // for(int k=0;k<40;k++){
            //     printf("%f, ",matrixCQ[k]);
            // }
        }
        {
            
            lrxigemm<float,8>(A_d,B_d,C_d,M,K,K,N,10, &cusolverH, &cublasH);
            hipMemcpy( matrixCQ,C_d, sizeof(float) * M * N, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            float R3 = get_Ferror<float>(matrixC,matrixCQ,M,N); 

            printf("%.7f\t",R3);
        }
        {
            //skxigemm<float,8>(A_d,B_d,C_d,M,K,K,N,1, &cusolverH, &cublasH);
            skxigemm_mem<float,8>(A_d,B_d,C_d,work,M,K,K,N,1, &cusolverH, &cublasH);
            hipMemcpy( matrixCQ,C_d, sizeof(float) * M * N, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            float R3 = get_Ferror<float>(matrixC,matrixCQ,M,N); 
            printf("%.7f\t",R3);
        }
        {
            //skxigemm<float,8>(A_d,B_d,C_d,M,K,K,N,1, &cusolverH, &cublasH);
            skxigemm_mem_fusion<float,8>(A_d,B_d,C_d,work,M,K,K,N,1, &cublasH);
            hipMemcpy( matrixCQ,C_d, sizeof(float) * M * N, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            float R3 = get_Ferror<float>(matrixC,matrixCQ,M,N); 
            printf("%.7f\n",R3);
        }
    }
    return;        
}


void performance_test(){
    int test_para[2048][5] = {

        // {2048,2048,2048,50},
        // {2048,2048,2048,40},
        // {2048,2048,2048,30},
        // {2048,2048,2048,20},
        // {2048,2048,2048,10},
        // {2048,2048,2048,1},
        // {16384,16384,16384,1},
        {2048,2048,2048,1},
        {4096,4096,4096,1},
        {4096*2,4096*2,4096*2,1},
        {4096*3,4096*3,4096*3,1},
        {4096*4,4096*4,4096*4,1},
        // {4096*5,4096*5,4096*5,1},
        // {16384,16384,16384,1},


        // {128,128,128,10,'k'},
        // {256,256,256,10,'k'},
        // {384,384,384,10,'k'},
        // {512,512,512,10,'k'},
        // {640,640,640,10,'k'},
        // {768,768,768,10,'k'},
        // {896,896,896,10,'k'},
        // {2048,2048,2048,10,'k'},
        // {1152,1152,1152,10,'k'},
        // {1280,1280,1280,10,'k'},
        // {1408,1408,1408,10,'k'},
        // {1536,1536,1536,10,'k'},
        // {1664,1664,1664,10,'k'},
        // {1792,1792,1792,10,'k'},
        // {1920,1920,1920,10,'k'},
        // {2048,2048,2048,10,'k'},
        // {2176,2176,2176,10,'k'},
        // {2304,2304,2304,10,'k'},
        // {2432,2432,2432,10,'k'},
        // {2560,2560,2560,10,'k'},
        // {2688,2688,2688,10,'k'},
        // {2816,2816,2816,10,'k'},
        // {2944,2944,2944,10,'k'},
        // {3072,3072,3072,10,'k'},
        // {3200,3200,3200,10,'k'},
        // {3328,3328,3328,10,'k'},
    }; 
  
    hipblasHandle_t cublasH = NULL;
    hipsolverHandle_t cusolverH = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    int max = 4096*4;
    float *matrixA = (float *)malloc(sizeof(float) * max*max);
    float *matrixB = (float *)malloc(sizeof(float) * max*max);
    float *matrixC = (float *)malloc(sizeof(float) * max*max);
    float *matrixCQ = (float *)malloc(sizeof(float) * max*max);
    float *matrixR = (float *)malloc(sizeof(float) * max*max);

    char * work;
    hipMalloc((char **)&work, sizeof(float) * (max*max*6+max*5));

    std::cout<<"M\tN\tK\trank\tSGEMM\t\torigin\t\tLrxigemm\tsketch\n";
    const int digit = 8;
    
    float *A_d, *B_d, *C_d;
    generate_matrix<float>(matrixA,max,max,'u');
    generate_matrix<float>(matrixB,max,max,'u');       
    for(int i=0;i<6;i++){
        

        int N=test_para[i][0],M=test_para[i][1],K=test_para[i][2];
        int rank =test_para[i][3];

        float alpha = 1.0, beta = 0.0;
        if(M==0) return;
        hipMalloc((float **)&A_d, sizeof(float) * M*K);
        hipMalloc((float **)&B_d, sizeof(float) * K*N);
        hipMalloc((float **)&C_d, sizeof(float) * M*N);
        hipMemcpy(A_d, matrixA, sizeof(float) * M*K, hipMemcpyHostToDevice);
        hipMemcpy(B_d, matrixB, sizeof(float) * K*N, hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        std::cout<<M<<"\t"<<N<<"\t"<<K<<"\t"<<rank<<"\t";

        //计算float和int矩阵乘法得到结果矩阵

        // xgemm(matrixA,matrixB,matrixC,M,K,K,N);

        {
            auto start = std::chrono::high_resolution_clock::now();
            cublas_gemm_rowmajor(
                &cublasH, A_d, B_d, C_d, M, K,
                K, N, alpha, beta);
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double, std::milli> diff = end - start;
            double time  = diff.count();
            printf("%.7lf\t",time);
            hipDeviceSynchronize();
        }



        {
            auto start = std::chrono::high_resolution_clock::now();
            xigemm_mem<float,8>(A_d,B_d,C_d,work,M,K,K,N);
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double, std::milli> diff = end - start;
            double time  = diff.count();
            printf("%.7lf\t",time);
        }
        {
            auto start = std::chrono::high_resolution_clock::now();
            //lrxigemm<float,8>(A_d,B_d,C_d,M,K,K,N,10, &cusolverH, &cublasH);
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double, std::milli> diff = end - start;
            double time  = diff.count();
            printf("%.7lf\t",time);
        }
        {
            skxigemm_mem<float,8>(A_d,B_d,C_d,work,128,128,128,128,1, &cusolverH, &cublasH);
            //skxigemm<float,8>(A_d,B_d,C_d,128,128,128,128,1, &cusolverH, &cublasH);
            auto start = std::chrono::high_resolution_clock::now();
            //skxigemm<float,8>(A_d,B_d,C_d,M,K,K,N,1, &cusolverH, &cublasH);
            skxigemm_mem<float,8>(A_d,B_d,C_d,work,M,K,K,N,1, &cusolverH, &cublasH);
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double, std::milli> diff = end - start;   
            double time  = diff.count();
            printf("%.7lf\t",time);
        }
        {
            skxigemm_mem_fusion<float,8>(A_d,B_d,C_d,work,128,128,128,128,1, &cublasH);
            //skxigemm<float,8>(A_d,B_d,C_d,128,128,128,128,1, &cusolverH, &cublasH);
            auto start = std::chrono::high_resolution_clock::now();
            //skxigemm<float,8>(A_d,B_d,C_d,M,K,K,N,1, &cusolverH, &cublasH);
            skxigemm_mem_fusion<float,8>(A_d,B_d,C_d,work,M,K,K,N,1, &cublasH);
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double, std::milli> diff = end - start;   
            double time  = diff.count();
            printf("%.7lf\n",time);
        }
        hipFree(A_d);hipFree(B_d);hipFree(C_d);

    }
    return;        
}



void nsys_perf_test(){
    int test_para[2048][5] = {
        {4096*4,4096*4,4096*4,1},
    }; 
  
    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    int max = 4096*4;
    float *matrixA = (float *)malloc(sizeof(float) * max*max);
    float *matrixB = (float *)malloc(sizeof(float) * max*max);
    float *matrixC = (float *)malloc(sizeof(float) * max*max);
    float *matrixCQ = (float *)malloc(sizeof(float) * max*max);
    float *matrixR = (float *)malloc(sizeof(float) * max*max);

    char * work;
    hipMalloc((char **)&work, sizeof(float) * (max*max*6+max*5));

    std::cout<<"M\tN\tK\trank\tSGEMM\t\torigin\t\tLrxigemm\tsketch\n";
    const int digit = 8;
    
    float *A_d, *B_d, *C_d;
    // generate_matrix<float>(matrixA,max,max,'u');
    // generate_matrix<float>(matrixB,max,max,'u');       
    for(int i=0;i<1;i++){
        

        int N=test_para[i][0],M=test_para[i][1],K=test_para[i][2];
        int rank =test_para[i][3];

        float alpha = 1.0, beta = 0.0;
        if(M==0) return;
        hipMalloc((float **)&A_d, sizeof(float) * M*K);
        hipMalloc((float **)&B_d, sizeof(float) * K*N);
        hipMalloc((float **)&C_d, sizeof(float) * M*N);
        // hipMemcpy(A_d, matrixA, sizeof(float) * M*K, hipMemcpyHostToDevice);
        // hipMemcpy(B_d, matrixB, sizeof(float) * K*N, hipMemcpyHostToDevice);
        hipDeviceSynchronize();

        //计算float和int矩阵乘法得到结果矩阵
        auto start = std::chrono::high_resolution_clock::now();
        skxigemm_mem_fusion<float,8>(A_d,B_d,C_d,work,M,K,K,N,1, &cublasH);
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> diff = end - start;   
        double time  = diff.count();
        printf("%.7lf\n",time);
    }
    return;        
}

void compare_print_test(){
    int test_para[2048][5] = {

        // {2048,2048,2048,50},
        // {2048,2048,2048,40},
        // {2048,2048,2048,30},
        // {2048,2048,2048,20},
        // {2048,2048,2048,10},
        {16,16,16,10,'k'},
    }; 
  
    hipblasHandle_t cublasH = NULL;
    hipsolverHandle_t cusolverH = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    int max = 8192;
    float *matrixA = (float *)malloc(sizeof(float) * max*max);
    float *matrixAR = (float *)malloc(sizeof(float) * max*max);
    float *matrixAR2 = (float *)malloc(sizeof(float) * max*max);
    std::cout<<"M\tN\tK\ttype\trank\torigin\t\tLrxigemm\tsketch\n";
    const int digit = 8;
    char * work;
    hipMalloc((char **)&work, sizeof(float) * (max*max*8+max*5));

    float *A_d, *AL_d, *AR_d, *A2_d, *RA_d, *RA2_d, *PA_d;
    int8_t *AI_d;
    for(int i=0;i<1;i++){
        

        int N=test_para[i][0],M=test_para[i][1],K=test_para[i][2];
        int rank =test_para[i][3];
        char type =test_para[i][4];

        float alpha = 1.0, beta = 0.0;

        if(i!=0) {
            if(N!=test_para[i-1][0]||M!=test_para[i-1][1]||K!=test_para[i-1][2]||type!=test_para[i-1][4]){
                generate_matrix<float>(matrixA,M,K,type);
            }
        } else {
            generate_matrix<float>(matrixA,M,K,type);
        }
        hipMalloc((float **)&A_d, sizeof(float) * M*N);
        hipMalloc((float **)&A2_d, sizeof(float) * M*N);
        hipMalloc((float **)&RA_d, sizeof(float) * M*N);
        hipMalloc((float **)&RA2_d, sizeof(float) * M*N);
        hipMalloc((float **)&PA_d, sizeof(float) * M*N);
        hipMalloc((int8_t **)&AI_d, sizeof(int8_t) * M*N);

        hipMalloc((float **)&AL_d, sizeof(float) * M);
        hipMalloc((float **)&AR_d, sizeof(float) * N);
        hipMemcpy(A_d, matrixA, sizeof(float) * M*N, hipMemcpyHostToDevice);
        hipDeviceSynchronize();




        const int max_int = (1<<(8-1)) - 1;
        float max_mA = cublas_absmax(&cublasH, A_d, M*N);
        float lambdaA = (float)max_int/max_mA;

        
        quantitize_getR_int8(A_d, AI_d, PA_d, RA_d, M, N, lambdaA);

        hipMemcpy(matrixAR , RA_d, sizeof(float) * M*N, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        printf("AR = \n");
        print_MatrixE(matrixAR,M,N);


        //计算float和int矩阵乘法得到结果矩阵
        hiprandGenerator_t gen;
        sketch_r1_re( RA_d, AL_d, AR_d,M, N, &gen, &cublasH);

        cublas_gemm_rowmajor(
            &cublasH, AL_d, AR_d, RA2_d,  M,  1,
            1,  N, alpha,  beta);
        hipDeviceSynchronize();
        hipMemcpy(matrixAR2 , RA2_d, sizeof(float) * M*N, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        printf("\n\n\nSKETCH-AR = \n");
        print_MatrixE(matrixAR2,M,N);
        alpha = -1.0;
        cublas_saxpy(RA_d, RA2_d ,alpha, M*N, cublasH);
        hipDeviceSynchronize();
        hipMemcpy(matrixAR2 , RA2_d, sizeof(float) * M*N, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        printf("\n\n\nSKETCH - RAR = \n");
        print_MatrixE(matrixAR2,M,N);

        float sum =0,sumabs = 0;
        for(int j=0;j<M*N;j++){
            sum += matrixAR2[j];
            sumabs  +=abs(matrixAR2[j]);
        }
        printf("\n\nsum=%.4f, sumabs=%.4f, avg = %.4f, avg_abs=%.4f\n",sum,sumabs,sum/(float(M*N)),sumabs/(float(M*N)));
        alpha = 1.0;
        //计算float和int矩阵乘法得到结果矩阵
        cusolver_rsvd_LR(M, N, RA_d, AL_d, AR_d, 1, &cusolverH);

        cublas_gemm_rowmajor(
            &cublasH, AL_d, AR_d, RA2_d,  M,  1,
            1,  N, alpha,  beta);
        hipDeviceSynchronize();
        hipMemcpy(matrixAR2 , RA2_d, sizeof(float) * M*N, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        printf("\n\n\nCUSOLVER - AR = \n");
        print_MatrixE(matrixAR2,M,N);

        alpha = -1.0;
        cublas_saxpy(RA_d, RA2_d ,alpha, M*N, cublasH);
        hipDeviceSynchronize();
        hipMemcpy(matrixAR2 , RA2_d, sizeof(float) * M*N, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        printf("\n\n\nCUSOLVER - RAR = \n");
        print_MatrixE(matrixAR2,M,N);

    }
    return;        
}

void xhgemm_acc(){
    int max = 1024;
    float *matrixA = (float *)malloc(sizeof(float) * max*max);
    float *matrixB = (float *)malloc(sizeof(float) * max*max);
    float *matrixC = (float *)malloc(sizeof(float) * max*max);
    float *matrixCQ = (float *)malloc(sizeof(float) * max*max);
    float *matrixR = (float *)malloc(sizeof(float) * max*max);

    int M=max , N=max, K = max;

    generate_matrix<float>(matrixA,M,K,'n');
    generate_matrix<float>(matrixB,K,N,'n');    

    xgemm(matrixA,matrixB,matrixC,M,K,K,N);

    float *A_d, *B_d, *C_d;
    hipMalloc((float **)&A_d, sizeof(float) * M*K);
    hipMalloc((float **)&B_d, sizeof(float) * K*N);
    hipMalloc((float **)&C_d, sizeof(float) * M*N);
    hipMemcpy(A_d, matrixA, sizeof(float) * M*K, hipMemcpyHostToDevice);
    hipMemcpy(B_d, matrixB, sizeof(float) * K*N, hipMemcpyHostToDevice);

    xigemm<float,8>(A_d,B_d,C_d,M,K,K,N);
    hipMemcpy( matrixCQ,C_d, sizeof(float) * M * N, hipMemcpyDeviceToHost);
    float R2 = get_Ferror<float>(matrixC,matrixCQ,M,N); 
    printf("%.7f\n",R2);

    hipblasHandle_t cublasH = NULL;
    hipsolverHandle_t cusolverH = NULL;
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    xhgemm<float>(A_d,B_d,C_d,M,K,K,N,&cublasH);
    hipMemcpy( matrixCQ,C_d, sizeof(float) * M * N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    float R3 = get_Ferror<float>(matrixC,matrixCQ,M,N); 

    printf("%.7f\n",R3);


    return ;
}

int main(){
    //skxigemm_acc();
    //curand_test();
    //sketch_acc_test();
    // performance_test();
    //precision_test();

    // nsys_perf_test();
    xhgemm_acc();
    //compare_print_test();
}